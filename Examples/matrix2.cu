#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>



__host__
void fill_vector(float *V, int len){
  float aux = 5.0;
  for (int i = 0; i < len; i++) {
    V[i] = ((float)rand() / (float)(RAND_MAX)) * aux ;
  }
}

__host__
void print(float *V, int len){
  for (int i = 0; i < len; i++) {
    printf("%.2f ", V[i]);
  }
  printf("\n");
}

__global__
void MatrixKernel(float* d_M, float* d_R, int n){
  //calculate row index of element
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < n) d_R[i] = 2 * d_M[i];
  return;
}


int main(){
  int n = 100;
  int size = n * sizeof(float);

  //CPU
  float *h_M = (float*)malloc(size);
  float *h_R = (float*)malloc(size);
  //GPU
  float *d_M, *d_R;
  hipMalloc((void**)&d_M, size);
  hipMalloc((void**)&d_R, size);

  //Fill Matrix
  fill_vector(h_M, size);

  //Copy from CPU to GPU
  hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);

  //Dimension kernel
  dim3 dimGrid(ceil(n/10.0), 1, 1);
  dim3 dimBlock(10,1,1);
  MatrixKernel<<<dimGrid, dimBlock>>>(d_M, d_R, n);

  hipMemcpy(h_R, d_R, size, hipMemcpyDeviceToHost);
  print(h_R, n);


  hipFree(d_M);
  hipFree(d_R);
  free(h_M);
  free(h_R);
  
  return 0;
}
